#include "hip/hip_runtime.h"
#include <stdexcept>
#include <vector>

#include "cudalaunch.hpp"
#include "field.hpp"
#include "fieldops.hpp"

__global__ void k_addFields(CuField y,
                            real a1,
                            const CuField x1,
                            real a2,
                            const CuField x2) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (!y.cellInGeometry(idx))
    return;
  for (int c = 0; c < y.ncomp; c++) {
    real term1 = a1 * x1.valueAt(idx, c % x1.ncomp);
    real term2 = a2 * x2.valueAt(idx, c % x2.ncomp);
    y.setValueInCell(idx, c, term1 + term2);
  }
}

__global__ void k_addFields(CuField y,
                            real3 a1,
                            const CuField x1,
                            real3 a2,
                            const CuField x2) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (!y.cellInGeometry(idx))
    return;

  real3 term1 = a1 * x1.FM_vectorAt(idx);
  real3 term2 = a2 * x2.FM_vectorAt(idx);
  y.setVectorInCell(idx, term1 + term2);
}

__global__ void k_addFields(CuField y,
                            real6 a1,
                            const CuField x1,
                            real6 a2,
                            const CuField x2) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (!y.cellInGeometry(idx))
    return;

  real6 term1 = a1 * x1.AFM_vectorAt(idx);
  real6 term2 = a2 * x2.AFM_vectorAt(idx);
  y.setVectorInCell(idx, term1 + term2);
}

inline void add(Field& y, real a1, const Field& x1, real a2, const Field& x2) {
  if (x1.system() != y.system() || x2.system() != y.system()) {
    throw std::invalid_argument(
        "Fields can not be added together because they belong to different "
        "systems)");
  }
  if ((x1.ncomp() != y.ncomp() || x1.ncomp() != y.ncomp()) ) {
    throw std::invalid_argument(
        "Fields can not be added because they do not have the same number of "
        "components");
  }
  int ncells = y.grid().ncells();
  hipLaunchByPtr(ncells, k_addFields, y.cu(), a1, x1.cu(), a2, x2.cu());
}

inline void add(Field& y,
                real3 a1,
                const Field& x1,
                real3 a2,
                const Field& x2) {
  if (x1.system() != y.system() || x2.system() != y.system()) {
    throw std::invalid_argument(
        "Fields can not be added together because they belong to different "
        "systems)");
  }
  if (x1.ncomp() != y.ncomp() || x1.ncomp() != y.ncomp()) {
    throw std::invalid_argument(
        "Fields can not be added because they do not have the same number of "
        "components");
  }
  if (x1.ncomp() != 3) {
    throw std::invalid_argument("Fields should have 3 components.");
  }
  int ncells = y.grid().ncells();
  hipLaunchByPtr(ncells, k_addFields, y.cu(), a1, x1.cu(), a2, x2.cu());
}

inline void add(Field& y,
                real6 a1,
                const Field& x1,
                real6 a2,
                const Field& x2) {
  if (x1.system() != y.system() || x2.system() != y.system()) {
    throw std::invalid_argument(
        "Fields can not be added together because they belong to different "
        "systems)");
  }
  if (x1.ncomp() != y.ncomp() || x1.ncomp() != y.ncomp()) {
    throw std::invalid_argument(
        "Fields can not be added because they do not have the same number of "
        "components");
  }
  if (x1.ncomp() != 6) {
    throw std::invalid_argument("Fields should have 6 components.");
  }
  int ncells = y.grid().ncells();
  hipLaunchByPtr(ncells, k_addFields, y.cu(), a1, x1.cu(), a2, x2.cu());
}

Field add(real a1, const Field& x1, real a2, const Field& x2) {
  Field y(x1.system(), x1.ncomp());
  add(y, a1, x1, a2, x2);
  return y;
}

Field add(const Field& x1, const Field& x2) {
  return add(1, x1, 1, x2);
}

void addTo(Field& y, real a, const Field& x) {
  add(y, 1, y, a, x);
}

void addTo(Field& y, real3 a, const Field& x) {
  real3 a0 = real3{1, 1, 1};
  add(y, a0, y, a, x);
}

void addTo(Field& y, real6 a, const Field& x) {
  real6 a0 = real6{1, 1, 1, 1, 1, 1};
  add(y, a0, y, a, x);
}

// TODO: this can be done much more efficient
Field add(std::vector<const Field*> x, std::vector<real> weights) {
  // TODO:: throw error if inputs are not compatible
  Field y = weights.at(0) * (*x.at(0));
  if (x.size() == 1) {
    return y;
  }

  for (int n = 1; n < x.size(); n++) {
    if (weights.at(n) != 0.0) {
      addTo(y, weights.at(n), *x.at(n));
    }
  }
  return y;
}

Field operator*(real a, const Field& x) {
  return add(0, x, a, x);
}

__global__ void k_addConstant(CuField y,
                              const CuField x,
                              real value,
                              int comp) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (!y.cellInGeometry(idx))
    return;
  y.setValueInCell(idx, comp, x.valueAt(idx, comp) + value);
}

__global__ void k_normalize(CuField dst, const CuField src) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (!dst.cellInGeometry(idx))
    return;

  int comp = src.ncomp;
  
  if (comp == 3) {
    real norm2 = 0.0;
    for (int c = 0; c < comp; c++) {
      real v = src.valueAt(idx, c);
      norm2 += v * v;
    }
    real invnorm = rsqrt(norm2);
    for (int c = 0; c < comp; c++) {
      real value = src.valueAt(idx, c) * invnorm;
      dst.setValueInCell(idx, c, value);
    }
  }
  else if (comp == 6) {
    real2 norm2 = real2{0., 0.};
    for (int c = 0; c < comp - 3; c++) {
      real v = src.valueAt(idx, c);
      real u = src.valueAt(idx, c + 3);
      norm2 += real2{v * v, u * u};
    }
    real2 invnorm = real2{rsqrt(norm2.x), rsqrt(norm2.y)};
    for (int c = 0; c < comp - 3; c++) {
      real vvalue = src.valueAt(idx, c) * invnorm.x;
      real uvalue = src.valueAt(idx, c + 3) * invnorm.y;
      dst.setValueInCell(idx, c, vvalue);
      dst.setValueInCell(idx, c + 3, uvalue);
    }
  }
}


Field normalized(const Field& src) {
  Field dst(Field(src.system(), src.ncomp()));
  hipLaunchByPtr(dst.grid().ncells(), k_normalize, dst.cu(), src.cu());
  return dst;
}

void normalize(Field& f) {
  hipLaunchByPtr(f.grid().ncells(), k_normalize, f.cu(), f.cu());
}

Field operator*(real3 a, const Field& x) {
  Field y(x.system(), x.ncomp());
  real3 a0 = real3{0, 0, 0};
  add(y, a0, x, a, x);
  return y;
}

Field operator*(real6 a, const Field& x) {
  Field y(x.system(), x.ncomp());
  real6 a0 = real6{0, 0, 0, 0, 0, 0};
  add(y, a0, x, a, x);
  return y;
}
