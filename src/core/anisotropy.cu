#include "hip/hip_runtime.h"
#include "anisotropy.hpp"
#include "ferromagnet.hpp"
#include "field.hpp"

AnisotropyField::AnisotropyField(Ferromagnet* ferromagnet)
    : FerromagnetQuantity(ferromagnet, 3, "anisotropy_field", "T") {}

__global__ void k_anisotropyField(CuField* hField,
                                  const CuField* mField,
                                  real3 anisU,
                                  real Ku1) {
  if (!hField->cellInGrid())
    return;
  real3 m = mField->cellVector();
  real3 h = Ku1 * anisU * m;
  hField->setCellVector(h);
}

void anisotropyField(Field* hField, const Ferromagnet* ferromagnet) {
  k_anisotropyField<<<1, hField->grid().ncells()>>>(
      hField->cu(), ferromagnet->magnetization()->cu(), ferromagnet->anisU,
      ferromagnet->ku1);
}

void AnisotropyField::evalIn(Field* result) const {
  anisotropyField(result, ferromagnet_);
}