#include "hip/hip_runtime.h"
#include <iostream>
#include <stdexcept>

#include "bufferpool.hpp"
#include "cudaerror.hpp"
#include "cudalaunch.hpp"
#include "cudastream.hpp"
#include "field.hpp"
#include "fieldops.hpp"
#include "fieldquantity.hpp"

Field::Field() : grid_({0, 0, 0}), ncomp_(0), devptr_devptrs_(nullptr) {}

Field::Field(Grid grid, int nComponents)
    : grid_(grid), ncomp_(nComponents), devptr_devptrs_(nullptr) {
  allocate();
}

Field::Field(const Field& other) : grid_(other.grid_), ncomp_(other.ncomp_) {
  allocate();
  copyFrom(&other);
}

// Move constructer
Field::Field(Field&& other) : grid_(other.grid_), ncomp_(other.ncomp_) {
  devptrs_ = other.devptrs_;
  devptr_devptrs_ = other.devptr_devptrs_;
  other.devptrs_.clear();
  other.devptr_devptrs_ = nullptr;
}

// Assignment operator
Field& Field::operator=(const Field& other) {
  if (this == &other) {
    return *this;
  }
  if (grid_ != other.grid_ || ncomp_ != other.ncomp_) {
    free();
    grid_ = other.grid_;
    ncomp_ = other.ncomp_;
    allocate();
  }
  copyFrom(&other);
  return *this;
}

// Evaluate quantity in this field
Field& Field::operator=(const FieldQuantity& q) {
  return operator=(q.eval());
}

// Move assignment operator
Field& Field::operator=(Field&& other) {
  grid_ = other.grid_;
  ncomp_ = other.ncomp_;
  devptrs_ = other.devptrs_;
  devptr_devptrs_ = other.devptr_devptrs_;
  other.devptrs_.clear();
  other.devptr_devptrs_ = nullptr;
  return *this;
}

Field::~Field() {
  free();
}

void Field::allocate() {
  if (ncomp_ == 0 || grid_.ncells() == 0)
    return;
  devptrs_.resize(ncomp_);
  for (auto& p : devptrs_) {
    p = bufferPool.allocate(grid_.ncells());
  }
  checkCudaError(hipMalloc((void**)&devptr_devptrs_, ncomp_ * sizeof(real*)));
  checkCudaError(hipMemcpyAsync(devptr_devptrs_, &devptrs_[0],
                                 ncomp_ * sizeof(real*), hipMemcpyHostToDevice,
                                 getCudaStream()));
}

void Field::free() {
  for (auto p : devptrs_) {
    bufferPool.recycle(p);
  }
  if (devptr_devptrs_)
    hipFree(devptr_devptrs_);
  devptr_devptrs_ = nullptr;
}

Grid Field::grid() const {
  return grid_;
}

int Field::ncomp() const {
  return ncomp_;
}

real* Field::devptr(int comp) const {
  return devptrs_.at(comp);
}

void Field::getData(real* buffer) const {
  for (int c = 0; c < ncomp_; c++) {
    real* bufferComponent = buffer + c * grid_.ncells();
    checkCudaError(hipMemcpyAsync(bufferComponent, devptrs_[c],
                                   grid_.ncells() * sizeof(real),
                                   hipMemcpyDeviceToHost, getCudaStream()));
  }
}

void Field::setData(real* buffer) {
  for (int c = 0; c < ncomp_; c++) {
    real* bufferComponent = buffer + c * grid_.ncells();
    checkCudaError(hipMemcpyAsync(devptrs_[c], bufferComponent,
                                   grid_.ncells() * sizeof(real),
                                   hipMemcpyHostToDevice, getCudaStream()));
  }
}

__global__ static void k_setComponent(CuField f, real value, int comp) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (!f.cellInGrid(idx))
    return;
  f.setValueInCell(idx, comp, value);
}

void Field::setUniformComponent(real value, int comp) {
  hipLaunchByPtr(grid_.ncells(), k_setComponent, cu(), value, comp);
}

void Field::makeZero() {
  for (int comp = 0; comp < ncomp_; comp++)
    setUniformComponent(0.0, comp);
}

void Field::copyFrom(const Field* src) {
  // TODO: throw error if field dimensions mismatch
  for (int c = 0; c < ncomp_; c++) {
    checkCudaError(hipMemcpyAsync(devptrs_[c], src->devptrs_[c],
                                   grid_.ncells() * sizeof(real),
                                   hipMemcpyDeviceToDevice, getCudaStream()));
  }
}

CuField Field::cu() const {
  return CuField{grid_, ncomp_, devptr_devptrs_};
}

void Field::operator+=(const Field& x) {
  addTo(*this, 1, x);
}