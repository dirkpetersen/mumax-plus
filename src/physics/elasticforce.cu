#include "hip/hip_runtime.h"
#include "elasticforce.hpp"
#include "cudalaunch.hpp"
#include "ferromagnet.hpp"
#include "field.hpp"
#include "parameter.hpp"


bool elasticForceAssuredZero(const Ferromagnet* magnet) {
  return ((!magnet->enableElastodynamics()) ||
          (magnet->c11.assuredZero() && magnet->c12.assuredZero() &&
           magnet->c44.assuredZero()));
}


/** Returns index of coo+relcoo if that is inside the geometry.
 * Otherwise it returns index of coo itself (assumed to be safe).
 * This mimics open boundary conditions.
*/
__device__ int coord2safeIndex(int3 coo, int3 relcoo,
                               const CuSystem& system, const Grid& mastergrid) {
  const Grid grid = system.grid;
  int3 coo_ = mastergrid.wrap(coo + relcoo);
  if (grid.cellInGrid(coo_)) {  // don't convert to index if outside grid!
    int idx_ = grid.coord2index(coo_);
    if (system.inGeometry(idx_))
      return idx_;
  }
  return grid.coord2index(coo);
}

/** Returns index of coo+relcoo1+relcoo2 if that is inside the geometry.
 * Otherwise returns index of coo+relcoo1 (first) or coo+relcoo2 (second)
 * if one of those is inside the geometry.
 * Or it returns index of coo itself (assumed to be safe).
 * This mimics open boundary conditions.
*/
__device__ int coord2safeIndex(int3 coo, int3 relcoo1, int3 relcoo2,
                               const CuSystem& system, const Grid& mastergrid) {
  const Grid grid = system.grid;
  int3 coo_[3] = {mastergrid.wrap(coo + relcoo1 + relcoo2),
                  mastergrid.wrap(coo + relcoo1), mastergrid.wrap(coo + relcoo2)};
  for (int i = 0; i < 3; i++) {
    if (grid.cellInGrid(coo_[i])) {  // don't convert to index if outside grid!
      int idx_ = grid.coord2index(coo_[i]);
      if (system.inGeometry(idx_))
        return idx_;
    }
  }
  return grid.coord2index(coo);
}

// ∂i(c ∂i(u))
// position index due to derivative, not component index!
// w = 1/cellsize
__device__ real doubleDerivative(real c_im1, real c_i, real c_ip1,
                                 real u_im1, real u_i, real u_ip1, real wi) {
  return (  harmonicMean(c_ip1, c_i) * (u_ip1 - u_i  )
          - harmonicMean(c_i, c_im1) * (u_i   - u_im1)) * (wi*wi);
}

// ∂j(c ∂i(u))
// ≈ ∂j(c)∂i(u) + c ∂j∂i(u)
// position index due to derivative, not component index!
// w = 1/cellsize
__device__ real mixedDerivative(real c_i_jm1, real c_i_j, real c_i_jp1,
                                real u_im1_j, real u_ip1_j,
                                real u_im1_jm1, real u_im1_jp1,
                                real u_ip1_jm1, real u_ip1_jp1,
                                real wi, real wj) {
  real f = (c_i_jp1 - c_i_jm1) * (u_ip1_j - u_im1_j);  // ~ ∂j(c)∂i(u)
  // optimal order of terms to minimize numerical noise for 2D materials (nz = 1)
  f += c_i_j * (u_ip1_jp1 - u_im1_jp1 - u_ip1_jm1 + u_im1_jm1);  // ~ c ∂j∂i(u)
  f *= 0.25 * wi * wj;
  return f;
}


// I tried to adhere to openBC==true using safeCoo
// similar to openBC in k_exchangeField (using continue)
// TODO: but need friction-free boundary conditions te be correct
__global__ void k_elasticForce(CuField fField,
                               const CuField uField,
                               const CuParameter c11,
                               const CuParameter c12,
                               const CuParameter c44,
                               const real3 w,
                               const Grid mastergrid) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const CuSystem system = fField.system;
  const Grid grid = system.grid;

  // When outside the geometry, set to zero and return early
  if (!system.inGeometry(idx)) {
    if (grid.cellInGrid(idx)) {
      fField.setVectorInCell(idx, real3{0, 0, 0});
    }
    return;
  }

  // array instead of real3 to get indexing [i]
  const real ws[3] = {w.x, w.y, w.z};
  const int gs[3] = {grid.size().x, grid.size().y, grid.size().z};
  const int3 ip1_arr[3] = {int3{ 1, 0, 0}, int3{0, 1, 0}, int3{0, 0, 1}};
  const int3 im1_arr[3] = {int3{-1, 0, 0}, int3{0,-1, 0}, int3{0, 0,-1}};
  const int3 coo = grid.index2coord(idx);

#pragma unroll  // TODO: check if faster with or without this unrolling
  for (int i = 0; i < 3; i++) {  // i is a {x, y, z} component/direction
    real f_i = 0;  // force component i

    // =============================================
    // f_i += ∂i(c11 ∂i(u_i))    so c11 with ∂²_i u_i
    // take u component i
    // translate all in direction i

    real ui = uField.valueAt(idx, i);
    real wi = ws[i];
    int3 im1 = im1_arr[i], ip1 = ip1_arr[i];  // transl in direction i
    int safeIdx_im1 = coord2safeIndex(coo, im1, system, mastergrid);
    int safeIdx_ip1 = coord2safeIndex(coo, ip1, system, mastergrid);
    if (gs[i] > 1)  // only derivative calculation if more than 1 cell
      f_i += doubleDerivative(c11.valueAt(safeIdx_im1),
                              c11.valueAt(idx),
                              c11.valueAt(safeIdx_ip1),
                              uField.valueAt(safeIdx_im1, i), ui,
                              uField.valueAt(safeIdx_ip1, i), wi);

#pragma unroll  // TODO: check if faster with or without this unrolling
    for (int j_=i+1; j_<i+3; j_++) {
      // j is one of the *other* {x, y, z} components/directions
      int j = j_;
      if (j > 2) {j -= 3;};

      if (gs[j] <= 1) continue;  // no derivative calculation if only 1 cell

      // translate in direction j
      real wj = ws[j];
      int3 jm1 = im1_arr[j], jp1 = ip1_arr[j];
      int safeIdx_jm1 = coord2safeIndex(coo, jm1, system, mastergrid);
      int safeIdx_jp1 = coord2safeIndex(coo, jp1, system, mastergrid);
      
      real c44_jm1 = c44.valueAt(safeIdx_jm1);
      real c44_    = c44.valueAt(idx);
      real c44_jp1 = c44.valueAt(safeIdx_jp1);

      // =============================================
      // f_i += ∂j(c44 ∂j(u_i))    so c44 with ∂²_j u_i
      // take u component i
      // translate all in j direction
      f_i += doubleDerivative(c44_jm1, c44_, c44_jp1,
                              uField.valueAt(safeIdx_jm1, i), ui,
                              uField.valueAt(safeIdx_jp1, i), wj);

      // ===========================================================
      // f_i += ∂j((c12+c44) ∂i(u_j))    so (c12+c44) with ∂_j∂_i u_j
      // translate c12+c44 in j direction
      // take u component j
      // translate u in both i and j directions

      if (gs[i] <= 1) continue;  // no derivative calculation if only 1 cell

      // translate in both i and j directions
      int safeIdx_im1_jm1 = coord2safeIndex(coo, im1, jm1, system, mastergrid);
      int safeIdx_im1_jp1 = coord2safeIndex(coo, im1, jp1, system, mastergrid);
      int safeIdx_ip1_jm1 = coord2safeIndex(coo, ip1, jm1, system, mastergrid);
      int safeIdx_ip1_jp1 = coord2safeIndex(coo, ip1, jp1, system, mastergrid);

      f_i += mixedDerivative(c12.valueAt(safeIdx_jm1) + c44_jm1,
                             c12.valueAt(idx) + c44_,
                             c12.valueAt(safeIdx_jp1) + c44_jp1,
                             uField.valueAt(safeIdx_im1, j),
                             uField.valueAt(safeIdx_ip1, j),
                             uField.valueAt(safeIdx_im1_jm1, j),
                             uField.valueAt(safeIdx_im1_jp1, j),
                             uField.valueAt(safeIdx_ip1_jm1, j),
                             uField.valueAt(safeIdx_ip1_jp1, j), wi, wj);
    }
    fField.setValueInCell(idx, i, f_i);
  }
}


Field evalElasticForce(const Ferromagnet* magnet) {

  Field fField(magnet->system(), 3);
  if (elasticForceAssuredZero(magnet)) {
    fField.makeZero();
    return fField;
  }

  int ncells = fField.grid().ncells();
  CuField uField = magnet->elasticDisplacement()->field().cu();
  CuParameter c11 = magnet->c11.cu();
  CuParameter c12 = magnet->c12.cu();
  CuParameter c44 = magnet->c44.cu();
  real3 w = 1 / magnet->cellsize();
  Grid mastergrid = magnet->world()->mastergrid();

  hipLaunchByPtr(ncells, k_elasticForce, fField.cu(), uField, c11, c12, c44, w,
             mastergrid);

  return fField;
}

FM_FieldQuantity elasticForceQuantity(const Ferromagnet* magnet) {
  return FM_FieldQuantity(magnet, evalElasticForce, 3, "elastic_force", "N/m3");
}
