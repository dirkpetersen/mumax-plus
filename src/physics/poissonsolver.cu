#include "hip/hip_runtime.h"
#include "cudalaunch.hpp"
#include "ferromagnetquantity.hpp"
#include "field.hpp"
#include "fieldops.hpp"
#include "linstepper.hpp"
#include "linsystem.hpp"
#include "poissonsolver.hpp"
#include "reduce.hpp"

PoissonSolver::PoissonSolver(const Ferromagnet* magnet)
    : magnet_(magnet),
      sys_(magnet->grid(), NNEAREST),
      pot_(magnet_->grid(), 1) {
  setMethod(Method::CONJUGATEGRADIENT);
}

__global__ static void k_construct(CuLinearSystem sys,
                                   const CuParameter pot,
                                   real3 cellsize) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;

  Grid grid = sys.grid;

  if (!grid.cellInGrid(idx))
    return;

  real vals[5] = {0, 0, 0, 0, 0};
  int colidx[5] = {idx, -1, -1, -1, -1};

  // cell coordinates of the neighbors
  int3 coo = grid.index2coord(idx);
  int3 neighbor[5];
  neighbor[0] = coo + int3{0, 0, 0};
  neighbor[1] = coo + int3{-1, 0, 0};
  neighbor[2] = coo + int3{1, 0, 0};
  neighbor[3] = coo + int3{0, -1, 0};
  neighbor[4] = coo + int3{0, 1, 0};

  if (!isnan(pot.valueAt(idx))) {
    vals[0] = 1.0;
    sys.b[idx] = pot.valueAt(idx);
  } else {
    for (int i = 1; i < 5; i++) {
      if (grid.cellInGrid(neighbor[i])) {
        vals[0] += 1.0;
        vals[i] -= 1.0;
        colidx[i] = grid.coord2index(neighbor[i]);
      }
    }
    sys.b[idx] = 0.0;
  }

  for (int c = 0; c < 5; c++) {
    sys.idx[c][idx] = colidx[c];
    sys.a[c][idx] = vals[c] / vals[0];
  }
}

void PoissonSolver::init() {
  nstep_ = 0;
  hipLaunchByPtr(sys_.grid().ncells(), k_construct, sys_.cu(),
             magnet_->appliedPotential.cu(), magnet_->cellsize());
  stepper_->restart();
}

Field PoissonSolver::solve() {
  init();

  while ((double)residualMaxNorm() > tol) {
    if (nstep_ > maxIterations && maxIterations >= 0) {
      break;
    }

    step();
    nstep_++;
  }

  return pot_;
}

void PoissonSolver::step() {
  stepper_->step();
}

void PoissonSolver::restart() {
  stepper_->restart();
}

void PoissonSolver::setMethod(Method method) {
  stepper_ = LinearSystemSolverStepper::create(&sys_, &pot_, method);
}

void PoissonSolver::setMethodByName(std::string methodName) {
  Method method = LinearSystemSolverStepper::getMethodByName(methodName);
  stepper_ = LinearSystemSolverStepper::create(&sys_, &pot_, method);
}

Field PoissonSolver::state() const {
  return pot_;
}

Field PoissonSolver::residual() const {
  return sys_.residual(pot_);
}

real PoissonSolver::residualMaxNorm() const {
  return maxAbsValue(sys_.residual(pot_));
}