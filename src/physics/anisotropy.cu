#include "hip/hip_runtime.h"
#include "anisotropy.hpp"
#include "cudalaunch.hpp"
#include "ferromagnet.hpp"
#include "field.hpp"
#include "parameter.hpp"
#include "world.hpp"

AnisotropyField::AnisotropyField(Ferromagnet* ferromagnet)
    : FerromagnetFieldQuantity(ferromagnet, 3, "anisotropy_field", "T") {}

__global__ void k_anisotropyField(CuField hField,
                                  const CuField mField,
                                  CuVectorParameter anisU,
                                  CuParameter Ku1,
                                  CuParameter msat) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (!hField.cellInGrid(idx))
    return;

  real3 u = normalized(anisU.vectorAt(idx));
  real3 m = mField.vectorAt(idx);
  real k = Ku1.valueAt(idx);
  real Ms = msat.valueAt(idx);

  real3 h = 2 * k * dot(m, u) * u / Ms;

  hField.setVectorInCell(idx, h);
}

void AnisotropyField::evalIn(Field* result) const {
  CuField h = result->cu();
  const CuField m = ferromagnet_->magnetization()->field()->cu();
  auto anisU = ferromagnet_->anisU.cu();
  auto ku1 = ferromagnet_->ku1.cu();
  auto msat = ferromagnet_->msat.cu();
  int ncells = ferromagnet_->grid().ncells();
  hipLaunchByPtr(ncells, k_anisotropyField, h, m, anisU, ku1, msat);
}

AnisotropyEnergyDensity::AnisotropyEnergyDensity(Ferromagnet* ferromagnet)
    : FerromagnetFieldQuantity(ferromagnet,
                               1,
                               "anisotropy_energy_density",
                               "J/m3") {}

__global__ void k_anisotropyEnergyDensity(CuField edens,
                                          CuField mField,
                                          CuVectorParameter anisU,
                                          CuParameter Ku1,
                                          CuParameter msat) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (!edens.cellInGrid(idx))
    return;

  if (msat.valueAt(idx) == 0.0) {
    edens.setValueInCell(idx, 0, 0.0);
    return;
  }

  real3 u = normalized(anisU.vectorAt(idx));
  real3 m = mField.vectorAt(idx);
  real k = Ku1.valueAt(idx);
  edens.setValueInCell(idx, 0, -k * dot(m, u) * dot(m, u));
}

void AnisotropyEnergyDensity::evalIn(Field* edens) const {
  CuField e = edens->cu();
  const CuField m = ferromagnet_->magnetization()->field()->cu();
  auto anisU = ferromagnet_->anisU.cu();
  auto ku1 = ferromagnet_->ku1.cu();
  auto msat = ferromagnet_->msat.cu();
  int ncells = ferromagnet_->grid().ncells();
  hipLaunchByPtr(ncells, k_anisotropyEnergyDensity, e, m, anisU, ku1, msat);
}

AnisotropyEnergy::AnisotropyEnergy(Ferromagnet* ferromagnet)
    : FerromagnetScalarQuantity(ferromagnet, "anisotropy_energy", "J") {}

real AnisotropyEnergy::eval() const {
  int ncells = ferromagnet_->grid().ncells();
  real edensAverage = ferromagnet_->anisotropyEnergyDensity()->average()[0];
  real cellVolume = ferromagnet_->world()->cellVolume();
  return ncells * edensAverage * cellVolume;
}