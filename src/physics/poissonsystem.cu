#include "hip/hip_runtime.h"
#include "cudalaunch.hpp"
#include "ferromagnetquantity.hpp"
#include "field.hpp"
#include "fieldops.hpp"
#include "linsystem.hpp"
#include "poissonsystem.hpp"
#include "stdint.h"

__global__ static void k_construct(CuLinearSystem sys,
                                   const CuParameter pot,
                                   real3 cellsize) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;

  Grid grid = sys.grid;

  if (!grid.cellInGrid(idx))
    return;

  real vals[5] = {0, 0, 0, 0, 0};
  int colidx[5] = {idx, -1, -1, -1, -1};

  // cell coordinates of the neighbors
  int3 coo = grid.index2coord(idx);
  int3 neighbor[5];
  neighbor[0] = coo + int3{0, 0, 0};
  neighbor[1] = coo + int3{-1, 0, 0};
  neighbor[2] = coo + int3{1, 0, 0};
  neighbor[3] = coo + int3{0, -1, 0};
  neighbor[4] = coo + int3{0, 1, 0};

  if (!isnan(pot.valueAt(idx))) {
    vals[0] = 1.0;
    sys.b[idx] = pot.valueAt(idx);
  } else {
    for (int i = 1; i < 5; i++) {
      if (grid.cellInGrid(neighbor[i])) {
        vals[0] += 1.0;
        vals[i] -= 1.0;
        colidx[i] = grid.coord2index(neighbor[i]);
      }
    }
    sys.b[idx] = 0.0;
  }

  for (int c = 0; c < 5; c++) {
    sys.idx[c][idx] = colidx[c];
    sys.a[c][idx] = vals[c] / vals[0];
  }
}

void PoissonSystem::construct() {
  hipLaunchByPtr(sys_.grid().ncells(), k_construct, sys_.cu(),
             magnet_->appliedPotential.cu(), magnet_->cellsize());
}

Field PoissonSystem::solve() {
  construct();
  Field x = Field(grid(), 1, 0.0);
  int nstep = 1000;
  for (int i = 0; i < nstep; i++) {
    Field r = sys_.residual(x);  // r = Ax-b
    x = add(1.0, x, -1.0, r);    // x = x-r
  }
  return x;
}