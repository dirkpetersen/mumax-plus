#include "hip/hip_runtime.h"
#include "ferromagnet.hpp"
#include "field.hpp"
#include "torque.hpp"

Torque::Torque(Ferromagnet* ferromagnet)
    : FerromagnetQuantity(ferromagnet, 3, "torque", "T") {}

__global__ void k_torque(CuField* torque,
                         const CuField* mField,
                         const CuField* hField,
                         real alpha) {
  if (!torque->cellInGrid())
    return;
  real3 m = mField->cellVector();
  real3 h = hField->cellVector();
  real3 mxh = cross(m, h);
  real3 mxmxh = cross(m, mxh);
  real3 t = -mxh - alpha * mxmxh;
  torque->setCellVector(t);
}

void Torque::evalIn(Field* torque) const {
  CuField * t = torque->cu();
  CuField * h = ferromagnet_->effectiveField()->eval()->cu();
  CuField * m = ferromagnet_->magnetization()->field()->cu();
  real alpha = ferromagnet_->alpha;
  int ncells = torque->grid().ncells();
  k_torque<<<1,ncells>>>(t, m, h, alpha);
}
