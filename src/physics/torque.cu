#include "hip/hip_runtime.h"
#include "cudalaunch.hpp"
#include "ferromagnet.hpp"
#include "field.hpp"
#include "torque.hpp"

Torque::Torque(Ferromagnet* ferromagnet)
    : FerromagnetQuantity(ferromagnet, 3, "torque", "T") {}

__global__ void k_torque(CuField torque,
                         CuField mField,
                         CuField hField,
                         real alpha) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (!torque.cellInGrid(idx))
    return;
  real3 m = mField.vectorAt(idx);
  real3 h = hField.vectorAt(idx);
  real3 mxh = cross(m, h);
  real3 mxmxh = cross(m, mxh);
  real3 t = -mxh - alpha * mxmxh;
  torque.setVectorInCell(idx, t);
}

void Torque::evalIn(Field* torque) const {
  auto h = ferromagnet_->effectiveField()->eval();
  auto m = ferromagnet_->magnetization()->field();
  real alpha = ferromagnet_->alpha;
  int ncells = torque->grid().ncells();
  hipLaunchByPtr(ncells, k_torque, torque->cu(), m->cu(), h.get()->cu(), alpha);
}
