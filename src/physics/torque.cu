#include "hip/hip_runtime.h"
#include "cudalaunch.hpp"
#include "ferromagnet.hpp"
#include "field.hpp"
#include "torque.hpp"
#include "constants.hpp"

Torque::Torque(Ferromagnet* ferromagnet)
    : FerromagnetQuantity(ferromagnet, 3, "torque", "T") {}

__global__ void k_torque(CuField torque,
                         CuField mField,
                         CuField hField,
                         real alpha) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (!torque.cellInGrid(idx))
    return;
  real3 m = mField.vectorAt(idx);
  real3 h = hField.vectorAt(idx);
  real3 mxh = cross(m, h);
  real3 mxmxh = cross(m, mxh);
  real3 t = -GAMMALL / (1 + alpha * alpha) * (mxh + alpha * mxmxh);
  torque.setVectorInCell(idx, t);
}

void Torque::evalIn(Field* torque) const {
  auto h = ferromagnet_->effectiveField()->eval();
  auto m = ferromagnet_->magnetization()->field();
  real alpha = ferromagnet_->alpha;
  int ncells = torque->grid().ncells();
  hipLaunchByPtr(ncells, k_torque, torque->cu(), m->cu(), h.get()->cu(), alpha);
}

RelaxTorque::RelaxTorque(Ferromagnet* ferromagnet)
    : FerromagnetQuantity(ferromagnet, 3, "damping_torque", "T") {}

__global__ void k_dampingtorque(CuField torque,
                                CuField mField,
                                CuField hField) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (!torque.cellInGrid(idx))
    return;
  real3 m = mField.vectorAt(idx);
  real3 h = hField.vectorAt(idx);
  real3 t = -GAMMALL*cross(m, cross(m, h));
  torque.setVectorInCell(idx, t);
}

void RelaxTorque::evalIn(Field* torque) const {
  auto h = ferromagnet_->effectiveField()->eval();
  auto m = ferromagnet_->magnetization()->field();
  int ncells = torque->grid().ncells();
  hipLaunchByPtr(ncells, k_dampingtorque, torque->cu(), m->cu(), h.get()->cu());
}
