#include "hip/hip_runtime.h"
#include "cudalaunch.hpp"
#include "ferromagnet.hpp"
#include "field.hpp"
#include "torque.hpp"
#include "constants.hpp"
#include "parameter.hpp"

Torque::Torque(Ferromagnet* ferromagnet)
    : FerromagnetQuantity(ferromagnet, 3, "torque", "T") {}

__global__ void k_torque(CuField torque,
                         CuField mField,
                         CuField hField,
                         CuParameter alpha) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (!torque.cellInGrid(idx))
    return;
  real3 m = mField.vectorAt(idx);
  real3 h = hField.vectorAt(idx);
  real  a = alpha.valueAt(idx);
  real3 mxh = cross(m, h);
  real3 mxmxh = cross(m, mxh);
  real3 t = -GAMMALL / (1 + a * a) * (mxh + a * mxmxh);
  torque.setVectorInCell(idx, t);
}

void Torque::evalIn(Field* torque) const {
  auto h = ferromagnet_->effectiveField()->eval();
  auto m = ferromagnet_->magnetization()->field();
  auto alpha = ferromagnet_->alpha.cu();
  int ncells = torque->grid().ncells();
  hipLaunchByPtr(ncells, k_torque, torque->cu(), m->cu(), h.get()->cu(), alpha);
}

RelaxTorque::RelaxTorque(Ferromagnet* ferromagnet)
    : FerromagnetQuantity(ferromagnet, 3, "damping_torque", "T") {}

__global__ void k_dampingtorque(CuField torque,
                                CuField mField,
                                CuField hField) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (!torque.cellInGrid(idx))
    return;
  real3 m = mField.vectorAt(idx);
  real3 h = hField.vectorAt(idx);
  real3 t = -GAMMALL*cross(m, cross(m, h));
  torque.setVectorInCell(idx, t);
}

void RelaxTorque::evalIn(Field* torque) const {
  auto h = ferromagnet_->effectiveField()->eval();
  auto m = ferromagnet_->magnetization()->field();
  int ncells = torque->grid().ncells();
  hipLaunchByPtr(ncells, k_dampingtorque, torque->cu(), m->cu(), h.get()->cu());
}
