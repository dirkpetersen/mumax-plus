#include "hip/hip_runtime.h"
#include <hiprand.h>

#include "constants.hpp"
#include "cudalaunch.hpp"
#include "ferromagnet.hpp"
#include "field.hpp"
#include "parameter.hpp"
#include "thermalnoise.hpp"
#include "world.hpp"

bool thermalNoiseAssuredZero(const Ferromagnet *magnet) {
  return magnet->temperature.assuredZero();
}

__global__ void k_thermalNoise(CuField noiseField,
                               CuParameter msat,
                               CuParameter alpha,
                               CuParameter temperature,
                               real preFactor) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (!noiseField.cellInGrid(idx))
    return;

  real Ms = msat.valueAt(idx);
  real T = temperature.valueAt(idx);
  real a = alpha.valueAt(idx);
  real3 noise = noiseField.vectorAt(idx);

  noise *= sqrt(preFactor * a * T / ((1 + a * a) * Ms));

  noiseField.setVectorInCell(idx, noise);
}

Field evalThermalNoise(const Ferromagnet * magnet) {
  Field noise(magnet->grid(),3);
  if (thermalNoiseAssuredZero(magnet)) {
    noise.makeZero();
    return noise;
  }

  int N = noise.grid().ncells();
  real mean = 0.0;
  real stddev = 1.0;
  for (int c = 0; c < 3; c++) {
    hiprandGenerateNormal(magnet->randomGenerator, noise.devptr(c), N, mean, stddev);
    // TODO: make this also work for real = double   (using
    // hiprandGenerateNormalDouble)
  }

  auto msat = magnet->msat.cu();
  auto alpha = magnet->alpha.cu();
  auto temperature = magnet->temperature.cu();
  real cellVolume = magnet->world()->cellVolume();
  real preFactor = 2 * KB * GAMMALL / cellVolume;
  hipLaunchByPtr(N, k_thermalNoise, noise.cu(), msat, alpha, temperature,
             preFactor);
  return noise;
}

FM_FieldQuantity thermalNoiseQuantity(const Ferromagnet * magnet) {
  return FM_FieldQuantity(magnet, evalThermalNoise, 3, "thermalNoise", "");
}
