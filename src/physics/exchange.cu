#include "hip/hip_runtime.h"
#include "cudalaunch.hpp"
#include "energy.hpp"
#include "exchange.hpp"
#include "ferromagnet.hpp"
#include "field.hpp"
#include "parameter.hpp"
#include "reduce.hpp"
#include "world.hpp"

bool exchangeAssuredZero(const Ferromagnet* magnet) {
  return magnet->aex.assuredZero() || magnet->msat.assuredZero();
}

__device__ static inline real harmonicMean(real a, real b) {
  if (a + b == 0.0)
    return 0.0;
  return 2 * a * b / (a + b);
}

__global__ void k_exchangeField(CuField hField,
                                const CuField mField,
                                const CuParameter aex,
                                const CuParameter msat,
                                const real3 cellsize) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (!hField.cellInGrid(idx))
    return;

  if (msat.valueAt(idx) == 0) {
    hField.setVectorInCell(idx, {0, 0, 0});
    return;
  }

  const int3 coo = hField.grid.index2coord(idx);
  const real3 m = mField.vectorAt(idx);
  const real a = aex.valueAt(idx);

  // accumulate exchange field in h for cell at idx, divide by msat at the end
  real3 h{0, 0, 0};

  int3 neighborRelativeCoordinates[6] = {int3{-1, 0, 0}, int3{0, -1, 0},
                                         int3{0, 0, -1}, int3{1, 0, 0},
                                         int3{0, 1, 0},  int3{0, 0, 1}};

  for (int3 relcoo : neighborRelativeCoordinates) {
    const int3 coo_ = coo + relcoo;
    const int idx_ = hField.grid.coord2index(coo_);

    if (hField.cellInGrid(coo_) && msat.valueAt(idx_) != 0) {
      real dr =
          cellsize.x * relcoo.x + cellsize.y * relcoo.y + cellsize.z * relcoo.z;
      real3 m_ = mField.vectorAt(idx_);
      real a_ = aex.valueAt(idx_);

      h += 2 * harmonicMean(a, a_) * (m_ - m) / (dr * dr);
    }
  }

  h /= msat.valueAt(idx);
  hField.setVectorInCell(idx, h);
}

Field evalExchangeField(const Ferromagnet* magnet) {
  Field hField(magnet->grid(), 3);
  if (exchangeAssuredZero(magnet)) {
    hField.makeZero();
    return hField;
  }
  hipLaunchByPtr(hField.grid().ncells(), k_exchangeField, hField.cu(),
             magnet->magnetization()->field().cu(), magnet->aex.cu(),
             magnet->msat.cu(), magnet->world()->cellsize());
  return hField;
}

Field evalExchangeEnergyDensity(const Ferromagnet* magnet) {
  if (exchangeAssuredZero(magnet))
    return Field(magnet->grid(), 1, 0.0);
  return evalEnergyDensity(magnet, evalExchangeField(magnet), 0.5);
}

real evalExchangeEnergy(const Ferromagnet* magnet) {
  if (exchangeAssuredZero(magnet))
    return 0;
  real edens = exchangeEnergyDensityQuantity(magnet).average()[0];
  int ncells = magnet->grid().ncells();
  real cellVolume = magnet->world()->cellVolume();
  return ncells * edens * cellVolume;
}

FM_FieldQuantity exchangeFieldQuantity(const Ferromagnet* magnet) {
  return FM_FieldQuantity(magnet, evalExchangeField, 3, "exchange_field", "T");
}

FM_FieldQuantity exchangeEnergyDensityQuantity(const Ferromagnet* magnet) {
  return FM_FieldQuantity(magnet, evalExchangeEnergyDensity, 1,
                          "exchange_energy_density", "J/m3");
}

FM_ScalarQuantity exchangeEnergyQuantity(const Ferromagnet* magnet) {
  return FM_ScalarQuantity(magnet, evalExchangeEnergy, "exchange_energy", "J");
}
