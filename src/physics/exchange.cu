#include "hip/hip_runtime.h"
#include "cudalaunch.hpp"
#include "exchange.hpp"
#include "ferromagnet.hpp"
#include "field.hpp"
#include "world.hpp"

ExchangeField::ExchangeField(Ferromagnet* ferromagnet)
    : FerromagnetQuantity(ferromagnet, 3, "exchange_field", "T") {}

__global__ void k_exchangeField(CuField hField,
                                CuField mField,
                                real aex,
                                real msat,
                                real3 cellsize) {

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (!hField.cellInGrid(idx))
    return;

  int3 coo = hField.grid.index2coord(idx);

  real3 m = mField.vectorAt(idx);
  real3 ddm{0, 0, 0};  // second derivative of m

  int3 neighborRelativeCoordinates[6] = {int3{-1, 0, 0}, int3{0, -1, 0},
                                         int3{0, 0, -1}, int3{1, 0, 0},
                                         int3{0, 1, 0},  int3{0, 0, 1}};

  for (int3 relcoo : neighborRelativeCoordinates) {
    int3 coo_ = coo + relcoo;
    real dr =
        cellsize.x * relcoo.x + cellsize.y * relcoo.y + cellsize.z * relcoo.z;
    if (hField.cellInGrid(coo_)) {
      real3 m_ = mField.vectorAt(coo_);
      ddm += (m_ - m) / (dr * dr);
    }
  }

  hField.setVectorInCell(idx, 2 * aex * ddm / msat);
}

void exchangeField(Field* hField, const Ferromagnet* ferromagnet) {
  hipLaunchByPtr(hField->grid().ncells(), k_exchangeField, hField->cu(),
             ferromagnet->magnetization()->field()->cu(), ferromagnet->aex,
             ferromagnet->msat, ferromagnet->world()->cellsize());
}

void ExchangeField::evalIn(Field* result) const {
  exchangeField(result, ferromagnet_);
}