#include "hip/hip_runtime.h"
#include "cudalaunch.hpp"
#include "demagkernel.hpp"
#include "field.hpp"
#include "grid.hpp"
#include "newell.hpp"

DemagKernel::DemagKernel(Grid grid, real3 cellsize)
    : cellsize_(cellsize), grid_(grid) {
  kernel_ = new Field(grid_, 6);
  compute();
}

DemagKernel::DemagKernel(Grid dst, Grid src, real3 cellsize)
    : cellsize_(cellsize), grid_(kernelGrid(dst, src)) {
  kernel_ = new Field(grid_, 6);
  compute();
}

DemagKernel::~DemagKernel() {
  delete kernel_;
}

__global__ void k_demagKernel(CuField kernel, real3 cellsize) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (!kernel.cellInGrid(idx))
    return;
  int3 coo = kernel.grid.index2coord(idx);
  kernel.setValueInCell(idx, 0, calcNewellNxx(coo, cellsize));
  kernel.setValueInCell(idx, 1, calcNewellNyy(coo, cellsize));
  kernel.setValueInCell(idx, 2, calcNewellNzz(coo, cellsize));
  kernel.setValueInCell(idx, 3, calcNewellNxy(coo, cellsize));
  kernel.setValueInCell(idx, 4, calcNewellNxz(coo, cellsize));
  kernel.setValueInCell(idx, 5, calcNewellNyz(coo, cellsize));
}

void DemagKernel::compute() {
  hipLaunchByPtr(grid_.ncells(), k_demagKernel, kernel_->cu(), cellsize_);
}

Grid DemagKernel::grid() const {
  return grid_;
}
real3 DemagKernel::cellsize() const {
  return cellsize_;
}

const Field* DemagKernel::field() const {
  return kernel_;
}

Grid DemagKernel::kernelGrid(Grid dst, Grid src) {
  int3 size = src.size() + dst.size() - int3{1, 1, 1};

  // add padding to get even dimensions if size is larger than 5
  // this will make the fft on this grid mush more efficient
  if (size.x > 5 && size.x % 2 == 1)
    size.x += 1;
  if (size.y > 5 && size.y % 2 == 1)
    size.y += 1;
  if (size.z > 5 && size.z % 2 == 1)
    size.z += 1;

  int3 origin = src.origin() + src.size() - dst.origin() - size;
  return Grid(size, origin);
}