#include "hip/hip_runtime.h"
#include "antiferromagnet.hpp"
#include "cudalaunch.hpp"
#include "dmi.hpp" // used for Neumann BC
#include "energy.hpp"
#include "afmexchange.hpp"
#include "ferromagnet.hpp"
#include "field.hpp"
#include "parameter.hpp"
#include "reduce.hpp"
#include "world.hpp"

bool afmExchangeAssuredZero(const Ferromagnet* magnet) {
  if (!magnet->isSublattice()) { return true; }

  return ((magnet->hostMagnet()->afmex_cell.assuredZero()
        && magnet->hostMagnet()->afmex_nn.assuredZero())
        || (magnet->msat.assuredZero()
        && magnet->hostMagnet()->getOtherSublattice(magnet)->msat.assuredZero()));
}

__global__ void k_afmExchangeField(CuField hField,
                                const CuField m1Field,
                                const CuField m2Field,
                                const CuParameter aex,
                                const CuParameter afmex_cell,
                                const CuParameter afmex_nn,
                                const CuParameter msat2,
                                const CuParameter latcon,
                                const real3 w,  // w = 1/cellsize^2
                                const Grid mastergrid,
                                const CuDmiTensor dmiTensor,
                                bool openBC) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const auto system = hField.system;

  // When outside the geometry, set to zero and return early
  if (!hField.cellInGeometry(idx)) {
    if (hField.cellInGrid(idx))
      hField.setVectorInCell(idx, real3{0, 0, 0});
    return;
  }

  const Grid grid = m2Field.system.grid;

  if (!grid.cellInGrid(idx))
    return;

  if (msat2.valueAt(idx) == 0) {
    hField.setVectorInCell(idx, real3{0, 0, 0});
    return;
  }

  const int3 coo = grid.index2coord(idx);
  const real3 m2 = m2Field.vectorAt(idx);
  const real a = aex.valueAt(idx);
  const real ac = afmex_cell.valueAt(idx);
  const real ann = afmex_nn.valueAt(idx);
  
  // If there is no FM-exchange at the boundary, open BC are assumed
  openBC = (a == 0) ? true : openBC;

  // accumulate exchange field in h for cell at idx, divide by msat at the end
  real3 h{0, 0, 0};

  // AFM exchange at idx
  const real l = latcon.valueAt(idx);
  h += 4 * ac * m2 / (l * l);
  
  // AFM exchange in NN cells
#pragma unroll
  for (int3 rel_coo : {int3{-1, 0, 0}, int3{1, 0, 0}, int3{0, -1, 0},
                            int3{0, 1, 0}, int3{0, 0, -1}, int3{0, 0, 1}}) {
    int3 coo_ = mastergrid.wrap(coo + rel_coo);

    if(!hField.cellInGeometry(coo_) && openBC)
      continue;
    
    const int idx_ = grid.coord2index(coo_);
    real delta = dot(rel_coo, system.cellsize);

    if(msat2.valueAt(idx_) != 0 || !openBC) {
      real3 m2_;
      real ann_;
      int3 normal = rel_coo * rel_coo;

      if(hField.cellInGeometry(coo_)) {
        m2_ = m2Field.vectorAt(idx_);
        ann_ = afmex_nn.valueAt(idx_);
      }
      else { // Neumann BC
      real3 Gamma1 = getGamma(dmiTensor, idx, normal, m1Field.vectorAt(idx));
      real fac = ann / (2 * a);
      if(fac == -1) {
        m2_ = m2 + Gamma1 / (4*a) * delta;
      }
      else {
        real3 Gamma2 = getGamma(dmiTensor, idx, normal, m2);
        m2_ = m2 + delta / (a * 2 * (1 - fac*fac)) * (Gamma2 - fac * Gamma1);
      }
      ann_ = ann;
      }
      h += harmonicMean(ann, ann_) * dot(normal, w) * (m2_ - m2);
    }
  }
  hField.setVectorInCell(idx, h / msat2.valueAt(idx));
}

Field evalAFMExchangeField(const Ferromagnet* magnet) {

  Field hField(magnet->system(), 3);
  
  if (afmExchangeAssuredZero(magnet)) {
    hField.makeZero();
    return hField;
  }

  real3 c = magnet->cellsize();
  real3 w = {1 / (c.x * c.x), 1 / (c.y * c.y), 1 / (c.z * c.z)};
  
  auto otherSub = magnet->hostMagnet()->getOtherSublattice(magnet);
  auto mag = magnet->magnetization()->field().cu();
  auto otherMag = otherSub->magnetization()->field().cu();
  auto msat2 = otherSub->msat.cu();
  auto aex = magnet->aex.cu();
  auto afmex_cell = magnet->hostMagnet()->afmex_cell.cu();
  auto afmex_nn = magnet->hostMagnet()->afmex_nn.cu();
  auto latcon = magnet->hostMagnet()->latcon.cu();
  auto BC = magnet->enableOpenBC;
  auto dmiTensor = magnet->dmiTensor.cu();

  hipLaunchByPtr(hField.grid().ncells(), k_afmExchangeField, hField.cu(),
            mag, otherMag, aex, afmex_cell, afmex_nn, msat2, latcon,
            w, magnet->world()->mastergrid(), dmiTensor, BC);
  return hField;
}

Field evalAFMExchangeEnergyDensity(const Ferromagnet* magnet) {
  if (afmExchangeAssuredZero(magnet))
    return Field(magnet->system(), 1, 0.0);
  return evalEnergyDensity(magnet, evalAFMExchangeField(magnet), 0.5);
}

real evalAFMExchangeEnergy(const Ferromagnet* magnet) {
  if (afmExchangeAssuredZero(magnet))
    return 0;
    
  real edens = AFMexchangeEnergyDensityQuantity(magnet).average()[0];

  int ncells = magnet->grid().ncells();
  real cellVolume = magnet->world()->cellVolume();
  return ncells * edens * cellVolume;
}

FM_FieldQuantity AFMexchangeFieldQuantity(const Ferromagnet* magnet) {
  return FM_FieldQuantity(magnet, evalAFMExchangeField, 3, "exchange_field", "T");
}

FM_FieldQuantity AFMexchangeEnergyDensityQuantity(const Ferromagnet* magnet) {
  return FM_FieldQuantity(magnet, evalAFMExchangeEnergyDensity, 1,
                          "afm_exchange_energy_density", "J/m3");
}

FM_ScalarQuantity AFMexchangeEnergyQuantity(const Ferromagnet* magnet) {
  return FM_ScalarQuantity(magnet, evalAFMExchangeEnergy, "afm_exchange_energy", "J");
}

__global__ void k_angle(CuField angleField,
                        const CuField mField1,
                        const CuField mField2,
                        const CuParameter afmex,
                        const CuParameter msat1,
                        const CuParameter msat2) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;

  // When outside the geometry, set to zero and return early
  if (!angleField.cellInGeometry(idx)) {
    if (angleField.cellInGrid(idx)) 
      angleField.setValueInCell(idx, 0, 0);
    return;
  }

  if (msat1.valueAt(idx) == 0 || msat2.valueAt(idx) == 0 || afmex.valueAt(idx) == 0) {
    angleField.setValueInCell(idx, 0, 0);
    return;
  }

  angleField.setValueInCell(idx, 0, acos(copysign(1.0, afmex.valueAt(idx))
                                            * dot(mField1.vectorAt(idx),
                                                  mField2.vectorAt(idx))));
}

Field evalAngleField(const Antiferromagnet* magnet) {
  Field angleField(magnet->system(), 1);

  hipLaunchByPtr(angleField.grid().ncells(), k_angle, angleField.cu(),
            magnet->sub1()->magnetization()->field().cu(),
            magnet->sub2()->magnetization()->field().cu(),
            magnet->afmex_cell.cu(),
            magnet->sub1()->msat.cu(), magnet->sub2()->msat.cu());
  return angleField;
}

real evalMaxAngle(const Antiferromagnet* magnet) {
  return maxAbsValue(evalAngleField(magnet));
}

AFM_FieldQuantity angleFieldQuantity(const Antiferromagnet* magnet) {
  return AFM_FieldQuantity(magnet, evalAngleField, 1, "angle_field", "");
}

AFM_ScalarQuantity maxAngle(const Antiferromagnet* magnet) {
  return AFM_ScalarQuantity(magnet, evalMaxAngle, "max_angle", "");
}