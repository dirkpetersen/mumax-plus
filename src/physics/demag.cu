#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>

#include <memory>
#include <vector>

#include "cudalaunch.hpp"
#include "demag.hpp"
#include "ferromagnet.hpp"
#include "field.hpp"
#include "math.h"
#include "world.hpp"

DemagField::DemagField(Ferromagnet* ferromagnet)
    : FerromagnetQuantity(ferromagnet, 3, "demag_field", "T"),
      demagkernel_(ferromagnet->grid(),
                   ferromagnet->grid(),
                   ferromagnet->world()->cellsize()) {}

#define __CUDAOP__ inline __device__ __host__

__CUDAOP__ hipDoubleComplex operator+(hipDoubleComplex a, hipDoubleComplex b) {
  return hipCadd(a, b);
}

__CUDAOP__ hipDoubleComplex operator*(hipDoubleComplex a, hipDoubleComplex b) {
  return hipCmul(a, b);
}

__global__ static void k_move(CuField out, CuField in) {
  if (!out.cellInGrid())
    return;
  int3 coo = out.grid.idx2coo(blockIdx.x * blockDim.x + threadIdx.x);
  int3 coo_ = coo - out.grid.origin() + in.grid.origin();
  for (int c = 0; c < out.ncomp; c++) {
    real value = in.cellInGrid(coo_) ? in.cellValue(coo_, c) : 0.0;
    out.setCellValue(c, value);
  }
}

__global__ static void k_unpad(CuField out, CuField in) {
  if (!out.cellInGrid())
    return;
  int3 coo = out.grid.idx2coo(blockIdx.x * blockDim.x + threadIdx.x);
  int3 coo_ = coo - out.grid.origin() + in.grid.origin() + in.grid.size() -
              out.grid.size();
  for (int c = 0; c < out.ncomp; c++) {
    out.setCellValue(c, in.cellValue(coo_, c));
  }
}

static void checkCufftResult(hipfftResult result) {
  if (result != HIPFFT_SUCCESS)
    std::cout << "whoops" << std::endl;
}

static void move(Field* out, const Field* in) {
  hipLaunchByPtr(out->grid().ncells(), k_move, out->cu(), in->cu());
}

__global__ static void k_apply_kernel(hipfftDoubleComplex* hx,
                                      hipfftDoubleComplex* hy,
                                      hipfftDoubleComplex* hz,
                                      hipfftDoubleComplex* mx,
                                      hipfftDoubleComplex* my,
                                      hipfftDoubleComplex* mz,
                                      hipfftDoubleComplex* kxx,
                                      hipfftDoubleComplex* kyy,
                                      hipfftDoubleComplex* kzz,
                                      hipfftDoubleComplex* kxy,
                                      hipfftDoubleComplex* kxz,
                                      hipfftDoubleComplex* kyz,
                                      hipfftDoubleComplex preFactor,
                                      int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n)
    return;

  hx[i] = preFactor * (kxx[i] * mx[i] + kxy[i] * my[i] + kxz[i] * mz[i]);
  hy[i] = preFactor * (kxy[i] * mx[i] + kyy[i] * my[i] + kyz[i] * mz[i]);
  hz[i] = preFactor * (kxz[i] * mx[i] + kyz[i] * my[i] + kzz[i] * mz[i]);
}

void convolution(Field* out, const Field* m, const Field* kern, real msat) {
  Grid grid = kern->grid();

  // put m on kernel grid
  std::unique_ptr<Field> mpad(new Field(grid, 3));
  move(mpad.get(), m);  // pad

  int3 size = grid.size();
  int3 fftSize{size.x / 2 + 1, size.y, size.z};

  // allocate temporary gpu buffers
  std::vector<hipfftDoubleComplex*> mfft(3);
  std::vector<hipfftDoubleComplex*> hfft(3);
  std::vector<hipfftDoubleComplex*> kfft(6);
  int ncells = fftSize.x * fftSize.y * fftSize.z;
  for (auto& p : mfft)
    hipMalloc((void**)&p, ncells * sizeof(hipfftDoubleComplex));
  for (auto& p : hfft)
    hipMalloc((void**)&p, ncells * sizeof(hipfftDoubleComplex));
  for (auto& p : kfft)
    hipMalloc((void**)&p, ncells * sizeof(hipfftDoubleComplex));

  hipfftHandle forwardPlan;
  hipfftHandle backwardPlan;
  checkCufftResult(
      hipfftPlan3d(&forwardPlan, size.z, size.y, size.x, HIPFFT_D2Z));
  checkCufftResult(
      hipfftPlan3d(&backwardPlan, size.z, size.y, size.x, HIPFFT_Z2D));

  // Forward fourier transforms
  for (int comp = 0; comp < 3; comp++)
    checkCufftResult(
        hipfftExecD2Z(forwardPlan, mpad->devptr(comp), mfft.at(comp)));
  for (int comp = 0; comp < 6; comp++)
    checkCufftResult(
        hipfftExecD2Z(forwardPlan, kern->devptr(comp), kfft.at(comp)));

  // apply kernel on m_fft
  const real MU0 = 4 * M_PI * 1e-7;  // TODO: move this to a general place
  hipfftDoubleComplex preFactor =
      make_hipDoubleComplex(-MU0 * msat / grid.ncells(), 0);
  hipLaunchByPtr(ncells, k_apply_kernel, hfft.at(0), hfft.at(1), hfft.at(2),
             mfft.at(0), mfft.at(1), mfft.at(2), kfft.at(0), kfft.at(1),
             kfft.at(2), kfft.at(3), kfft.at(4), kfft.at(5), preFactor, ncells);

  // backward fourier transfrom
  for (int comp = 0; comp < 3; comp++)
    checkCufftResult(
        hipfftExecZ2D(backwardPlan, hfft.at(comp), mpad->devptr(comp)));

  // clean up temporary gpu memory buffers
  for (auto p : mfft)
    hipFree(p);
  for (auto p : kfft)
    hipFree(p);
  for (auto p : hfft)
    hipFree(p);
  checkCufftResult(hipfftDestroy(forwardPlan));
  checkCufftResult(hipfftDestroy(backwardPlan));

  hipLaunchByPtr(out->grid().ncells(), k_unpad, out->cu(), mpad->cu());  // unpad
}
__global__ void k_demagfield(CuField hField,
                             CuField mField,
                             CuField kernel,
                             real msat) {
  if (!hField.cellInGrid())
    return;

  real3 h{0, 0, 0};

  Grid g = mField.grid;
  int3 dstcoo = g.idx2coo(blockIdx.x * blockDim.x + threadIdx.x);

  for (int i = 0; i < g.ncells(); i++) {
    int3 srccoo = g.idx2coo(i);
    int3 dist = dstcoo - srccoo;

    real3 m = mField.cellVector(i);

    real nxx = kernel.cellValue(dist, 0);
    real nyy = kernel.cellValue(dist, 1);
    real nzz = kernel.cellValue(dist, 2);
    real nxy = kernel.cellValue(dist, 3);
    real nxz = kernel.cellValue(dist, 4);
    real nyz = kernel.cellValue(dist, 5);

    h.x -= nxx * m.x + nxy * m.y + nxz * m.z;
    h.y -= nxy * m.x + nyy * m.y + nyz * m.z;
    h.z -= nxz * m.x + nyz * m.y + nzz * m.z;
  }
  const real MU0 = 4 * M_PI * 1e-7;  // TODO: move this to a general place
  hField.setCellVector(msat * MU0 * h);
}

void DemagField::evalIn(Field* result) const {
  const Field* m = ferromagnet_->magnetization()->field();
  const Field* kernel = demagkernel_.field();
  real msat = ferromagnet_->msat;
  int ncells = result->grid().ncells();

  convolution(result, m, kernel, msat);

  //// brute method
  // hipLaunchByPtr(ncells, k_demagfield, result->cu(), m->cu(), kernel->cu(),
  // msat);
}