#include "hip/hip_runtime.h"
#include <memory>

#include "antiferromagnet.hpp"
#include "constants.hpp"
#include "cudalaunch.hpp"
#include "ferromagnet.hpp"
#include "magnet.hpp"
#include "field.hpp"
#include "fieldops.hpp"
#include "grid.hpp"
#include "parameter.hpp"
#include "strayfieldbrute.hpp"
#include "system.hpp"
#include "totalmag.hpp"

__global__ void k_demagfield(CuField hField,
                             const CuField mField,
                             const CuField kernel,
                             const CuParameter msat) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  // When outside the geometry of destiny field, set to zero and return
  // early
  if (!hField.cellInGeometry(idx)) {
    if (hField.cellInGrid(idx))
      hField.setVectorInCell(idx, real3{0, 0, 0});
    return;
  }

  int3 dstcoo = hField.system.grid.index2coord(idx);
  real3 h{0, 0, 0};

  for (int i = 0; i < mField.system.grid.ncells(); i++) {
    if (!mField.cellInGeometry(i))
      continue;

    int3 srccoo = mField.system.grid.index2coord(i);
    int3 r = dstcoo - srccoo;
    real nxx = kernel.valueAt(r, 0);
    real nyy = kernel.valueAt(r, 1);
    real nzz = kernel.valueAt(r, 2);
    real nxy = kernel.valueAt(r, 3);
    real nxz = kernel.valueAt(r, 4);
    real nyz = kernel.valueAt(r, 5);
    
    real3 M = msat.valueAt(i) * mField.FM_vectorAt(i);

    h.x -= nxx * M.x + nxy * M.y + nxz * M.z;
    h.y -= nxy * M.x + nyy * M.y + nyz * M.z;
    h.z -= nxz * M.x + nyz * M.y + nzz * M.z;
  }
  
  hField.setVectorInCell(idx, MU0 * h);
}

StrayFieldBruteExecutor::StrayFieldBruteExecutor(
    const Magnet* magnet,
    std::shared_ptr<const System> system)
    : StrayFieldExecutor(magnet, system),
      kernel_(system->grid(), magnet_->grid(), magnet_->world()) {}

Field StrayFieldBruteExecutor::exec() const {
  
  Field h(system_, 3);
  int ncells = h.grid().ncells();

  if(const Ferromagnet* mag = dynamic_cast<const Ferromagnet*>(magnet_)) {
    auto m = mag->magnetization()->field().cu();
    hipLaunchByPtr(ncells, k_demagfield, h.cu(), m, kernel_.field().cu(),
              mag->msat.cu());
  }
  else if (const Antiferromagnet* mag = dynamic_cast<const Antiferromagnet*>(magnet_)) {
    auto m = add(mag->sub1()->magnetization()->field(), mag->sub2()->magnetization()->field());

    hipLaunchByPtr(ncells, k_demagfield, h.cu(), m.cu(), kernel_.field().cu(),
              mag->sub1()->msat.cu());
  }
  return h;
}
