#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>

#include <memory>
#include <vector>

#include "cudalaunch.hpp"
#include "demagconvolution.hpp"
#include "demagkernel.hpp"
#include "field.hpp"
#include "timer.hpp"

#define __CUDAOP__ inline __device__ __host__

__CUDAOP__ complex operator+(complex a, complex b) {
  return hipCaddf(a, b);
}

__CUDAOP__ complex operator*(complex a, complex b) {
  return hipCmulf(a, b);
}

__global__ static void k_pad(CuField out, CuField in) {
  int outIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if (outIdx >= out.grid.ncells())
    return;

  int3 outCoo = out.grid.index2coord(outIdx);
  int3 inCoo = outCoo - out.grid.origin() + in.grid.origin();
  int inIdx = in.grid.coord2index(inCoo);

  for (int c = 0; c < out.ncomp; c++) {
    real value = in.grid.cellInGrid(inCoo) ? in.ptrs[c][inIdx] : 0.0;
    out.ptrs[c][outIdx] = value;
  }
}

__device__ inline int3 idx2coo(int idx, int3 gridSize) {
  return {idx % gridSize.x, (idx / gridSize.x) % gridSize.y,
          idx / (gridSize.x * gridSize.y)};
}

__global__ static void k_unpad(CuField out, CuField in) {
  int outIdx = blockIdx.x * blockDim.x + threadIdx.x;

  if (outIdx >= out.grid.ncells())
    return;

  int3 outCoo = out.grid.index2coord(outIdx);

  int3 cooIn = outCoo + in.grid.size() - out.grid.size() + in.grid.origin() -
               out.grid.origin();

  int inIdx = in.grid.coord2index(cooIn);

  for (int c = 0; c < out.ncomp; c++) {
    out.ptrs[c][outIdx] = in.ptrs[c][inIdx];
  }
}

static void checkCufftResult(hipfftResult result) {
  if (result != HIPFFT_SUCCESS)
    throw std::runtime_error("cufft error in demag convolution");
}

__global__ static void k_apply_kernel(complex* hx,
                                      complex* hy,
                                      complex* hz,
                                      complex* mx,
                                      complex* my,
                                      complex* mz,
                                      complex* kxx,
                                      complex* kyy,
                                      complex* kzz,
                                      complex* kxy,
                                      complex* kxz,
                                      complex* kyz,
                                      complex preFactor,
                                      int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n)
    return;

  hx[i] = preFactor * (kxx[i] * mx[i] + kxy[i] * my[i] + kxz[i] * mz[i]);
  hy[i] = preFactor * (kxy[i] * mx[i] + kyy[i] * my[i] + kyz[i] * mz[i]);
  hz[i] = preFactor * (kxz[i] * mx[i] + kyz[i] * my[i] + kzz[i] * mz[i]);
}

__global__ static void k_apply_kernel_2d(complex* hx,
                                         complex* hy,
                                         complex* hz,
                                         complex* mx,
                                         complex* my,
                                         complex* mz,
                                         complex* kxx,
                                         complex* kyy,
                                         complex* kzz,
                                         complex* kxy,
                                         complex preFactor,
                                         int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n)
    return;

  hx[i] = preFactor * (kxx[i] * mx[i] + kxy[i] * my[i]);
  hy[i] = preFactor * (kxy[i] * mx[i] + kyy[i] * my[i]);
  hz[i] = preFactor * kzz[i] * mz[i];
}

DemagConvolution::DemagConvolution(Grid grid, real3 cellsize)
    : grid_(grid),
      cellsize_(cellsize),
      kernel_(grid, grid, cellsize),
      kfft(6),
      hfft(3),
      mfft(3) {
  int3 size = kernel_.grid().size();
  fftSize = {size.x / 2 + 1, size.y, size.z};
  int ncells = fftSize.x * fftSize.y * fftSize.z;

  for (auto& p : kfft)
    hipMalloc((void**)&p, ncells * sizeof(complex));
  for (auto& p : mfft)
    hipMalloc((void**)&p, ncells * sizeof(complex));
  for (auto& p : hfft)
    hipMalloc((void**)&p, ncells * sizeof(complex));

  checkCufftResult(
      hipfftPlan3d(&forwardPlan, size.z, size.y, size.x, HIPFFT_R2C));
  checkCufftResult(
      hipfftPlan3d(&backwardPlan, size.z, size.y, size.x, HIPFFT_C2R));

  hipfftSetStream(forwardPlan, getCudaStream());
  hipfftSetStream(backwardPlan, getCudaStream());

  for (int comp = 0; comp < 6; comp++)
    checkCufftResult(hipfftExecR2C(forwardPlan, kernel_.field()->devptr(comp),
                                  kfft.at(comp)));
}

DemagConvolution::~DemagConvolution() {
  for (auto p : mfft)
    hipFree(p);
  for (auto p : kfft)
    hipFree(p);
  for (auto p : hfft)
    hipFree(p);

  checkCufftResult(hipfftDestroy(forwardPlan));
  checkCufftResult(hipfftDestroy(backwardPlan));
}

void DemagConvolution::exec(Field* h, const Field* m, real msat) const {

  std::unique_ptr<Field> mpad(new Field(kernel_.grid(), 3));
  hipLaunchByPtr(mpad->grid().ncells(), k_pad, mpad->cu(), m->cu());

  // Forward fourier transforms
  for (int comp = 0; comp < 3; comp++)
    checkCufftResult(
        hipfftExecR2C(forwardPlan, mpad->devptr(comp), mfft.at(comp)));

  // apply kernel on m_fft
  int ncells = fftSize.x * fftSize.y * fftSize.z;
  const real MU0 = 4 * M_PI * 1e-7;  // TODO: move this to a general place
  complex preFactor{-MU0 * msat / kernel_.grid().ncells(), 0};
  if (fftSize.z == 1) {
    hipLaunchByPtr(ncells, k_apply_kernel_2d, hfft.at(0), hfft.at(1), hfft.at(2),
               mfft.at(0), mfft.at(1), mfft.at(2), kfft.at(0), kfft.at(1),
               kfft.at(2), kfft.at(3), preFactor, ncells);

  } else {
    hipLaunchByPtr(ncells, k_apply_kernel, hfft.at(0), hfft.at(1), hfft.at(2),
               mfft.at(0), mfft.at(1), mfft.at(2), kfft.at(0), kfft.at(1),
               kfft.at(2), kfft.at(3), kfft.at(4), kfft.at(5), preFactor,
               ncells);
  }

  // backward fourier transfrom
  for (int comp = 0; comp < 3; comp++)
    checkCufftResult(
        hipfftExecC2R(backwardPlan, hfft.at(comp), mpad->devptr(comp)));

  // unpad
  hipLaunchByPtr(h->grid().ncells(), k_unpad, h->cu(), mpad->cu());
}