#include "cudastream.hpp"
#include "cudaerror.hpp"

hipStream_t stream0;

hipStream_t getCudaStream() {
  if (!stream0)
    hipStreamCreate(&stream0);
  return stream0;
}

// Multi-GPU utility functions
void initializeMultiGpu() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    
    if (deviceCount > 1) {
        // Enable peer access between all GPUs
        for (int i = 0; i < deviceCount; i++) {
            hipSetDevice(i);
            for (int j = 0; j < deviceCount; j++) {
                if (i != j) {
                    int canAccess;
                    hipDeviceCanAccessPeer(&canAccess, i, j);
                    if (canAccess) {
                        hipDeviceEnablePeerAccess(j, 0);
                    }
                }
            }
        }
        hipSetDevice(0); // Reset to default device
    }
}

int getNumDevices() {
    int count;
    hipGetDeviceCount(&count);
    return count;
}

void setCurrentDevice(int deviceId) {
    hipSetDevice(deviceId);
}

void synchronizeAllDevices() {
    int deviceCount = getNumDevices();
    for (int i = 0; i < deviceCount; i++) {
        hipSetDevice(i);
        hipDeviceSynchronize();
    }
    hipSetDevice(0);
}
