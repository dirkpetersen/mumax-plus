#include "hip/hip_runtime.h"
#include <stdexcept>

#include "cudalaunch.hpp"
#include "vec.hpp"

__global__ void k_add(real* y, real a1, real* x1, real a2, real* x2, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= N)
    return;
  y[idx] = a1 * x1[idx] + a2 * x2[idx];
}

GVec add(real a1, const GVec& x1, real a2, const GVec& x2) {
  if (x1.size() != x2.size()) {
    throw std::invalid_argument(
        "Arrays can not be added together because their dimensions do not "
        "match");
  }
  int N = x1.size();
  GVec y(N);
  hipLaunchByPtr(N, k_add, y.get(), a1, x1.get(), a2, x2.get(), N);
  return y;
}

GVec add(const GVec& x1, const GVec& x2) {
  return add(1, x1, 1, x2);
}

__global__ void k_maxAbsValue(real* result, real* x, int N) {
  // Reduce to a block
  __shared__ real sdata[BLOCKDIM];
  int tid = threadIdx.x;
  real threadValue = 0.0;
  for (int i = tid; i < N; i += BLOCKDIM) {
    real value = abs(x[i]);
    threadValue = value > threadValue ? value : threadValue;
  }
  sdata[tid] = threadValue;
  __syncthreads();

  // Reduce the block
  for (unsigned int s = BLOCKDIM / 2; s > 0; s >>= 1) {
    if (tid < s)
      if (sdata[tid + s] > sdata[tid])
        sdata[tid] = sdata[tid + s];
    __syncthreads();
  }

  // Set the result
  if (tid == 0)
    *result = sdata[0];
}

real maxAbsValue(const GVec& x) {
  if (x.size() == 0)
    return 0.0;

  real* d_result = (real*)bufferPool.allocate(sizeof(real));
  cudaLaunchReductionKernel(k_maxAbsValue, d_result, x.get(), (int)x.size());

  // copy the result to the host and return
  real result;
  checkCudaError(hipMemcpyAsync(&result, d_result, 1 * sizeof(real),
                                 hipMemcpyDeviceToHost, getCudaStream()));
  bufferPool.recycle((void**)&d_result);
  return result;
}

__global__ void k_dotSum(real* result, real* f, real* g, int N) {
  __shared__ real sdata[BLOCKDIM];
  int tid = threadIdx.x;

  real threadValue = 0.0;
  for (int i = tid; i < N; i += BLOCKDIM)
    threadValue += f[i] * g[i];

  sdata[tid] = threadValue;
  __syncthreads();

  // Reduce the block
  for (unsigned int s = BLOCKDIM / 2; s > 0; s >>= 1) {
    if (tid < s)
      sdata[tid] += sdata[tid + s];
    __syncthreads();
  }

  // Set the result
  if (tid == 0)
    *result = sdata[0];
}

real dotSum(const GVec& x1, const GVec& x2) {
  if (x1.size() != x2.size()) {
    throw std::invalid_argument(
        "Arrays can not be added together because their dimensions do not "
        "match");
  }

  if (x1.size() == 0)
    return 0.0;

  real* d_result = (real*)bufferPool.allocate(sizeof(real));
  cudaLaunchReductionKernel(k_dotSum, d_result, x1.get(), x2.get(),
                            (int)x1.size());
  // copy the result to the host and return
  real result;
  checkCudaError(hipMemcpyAsync(&result, d_result, sizeof(real),
                                 hipMemcpyDeviceToHost, getCudaStream()));
  bufferPool.recycle((void**)&d_result);
  return result;
}